#include "hip/hip_runtime.h"
//kernelPBO.cu (Rob Farber)
#include <stdio.h>

#include <cstdlib>
#include <iostream>
#include <limits>

#include "BVH.h"

#define GPU __host__ __device__
#include "data.h"

#define WIDTH 1024
#define HEIGHT 1024
#define TILE_WIDTH 16
#define THRESHOLD 0.001f
#define MAX_SPHERES 2048
#define VIEW_ANGLE (3.14159f / 3.f)
#define RADIUS .3f


#define USE_CONSTANT_MEMORY
#define USE_BVH
//#define USE_LINE_POINTS
#define USE_BUNNY
//#define USE_SPHERE_TEST
#define ASSUME_ALL_CONST


#ifdef ASSUME_ALL_CONST
 #define USE_CONSTANT_MEMORY
#else
 #define USE_SOME_CONSTANT_MEMORY
#endif
 
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line) {
   if(err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(EXIT_FAILURE);
   }
}

namespace cuda {

#ifdef USE_CONSTANT_MEMORY
__constant__ data::Sphere constSpheres[MAX_SPHERES];
#endif

// Calculates if and where a ray intersects a sphere
__device__ bool SphereIntersectionTest(data::Sphere s, data::Ray r, data::Hit *h) {
   data::vec3 o = r.start - s.center;
   data::vec3 dir = r.dir.normalize();

   float b = 2.f * dir.dot(o);
   float c = o.dot(o) - s.radius * s.radius;

   float determinant = b * b - 4 * c;   
   float dist = sqrtf(determinant) / 2.0f;

   if (determinant < THRESHOLD) {
      return false;
   } else {
      float d1 = -b / 2.f + dist,  d2 = -b / 2.f - dist;
      h->depth = (d1 < THRESHOLD || d2 < THRESHOLD) ? max(d1, d2) : min(d1, d2);
      h->position = r.start + dir * h->depth;
      h->normal = (h->position - s.center).normalize();
      return d1 >= THRESHOLD || d2 >= THRESHOLD;
   }
}

#ifdef USE_SPHERE_TEST
__device__ void CastRayBVH(data::Ray &r, data::Sphere *spheres, data::Hit *sh, int *t) {
#else
__device__ void CastRayBVH(data::Ray &r, data::Sphere *spheres, data::Hit *sh) {
#endif
   data::Hit h;
   int stack[20], *sp = &stack[1];
   stack[0] = 0, stack[1] = 1;
   while(sp >= &stack[1]) {
#ifdef USE_SPHERE_TEST
      (*t)++;
#endif
      bool hit = false;
#if defined(USE_CONSTANT_MEMORY) and defined(USE_SOME_CONSTANT_MEMORY)
      if(*sp < MAX_SPHERES) {
         hit = SphereIntersectionTest(constSpheres[*sp], r, &h);
      } else {
         hit = SphereIntersectionTest(spheres[*sp], r, &h);
      }    
#else
      hit = SphereIntersectionTest(spheres[*sp], r, &h);
#endif
      if(hit){
         if (spheres[*sp].draw && h.depth < sh->depth) {
            *sh = h;
            while (*sp % 2) sp--;
            (*sp)++;
         } else {
            int first_child = (*sp) << 1; 
            (*++sp) = first_child;
         }
      } else {
         while (*sp % 2) sp--;
         (*sp)++;
      }
   }
}

#ifdef USE_SPHERE_TEST
__device__ void CastRayLinear(data::Ray &r, data::Sphere *spheres, int numSpheres, data::Hit *sh, int *t) {
#else
__device__ void CastRayLinear(data::Ray &r, data::Sphere *spheres, int numSpheres, data::Hit *sh) {
#endif
   data::Hit h;
   for(int i = 0; i < numSpheres; i++) {
#ifdef USE_SPHERE_TEST
      (*t)++;
#endif
      bool hit = false;
#if defined(USE_CONSTANT_MEMORY) and defined(USE_SOME_CONSTANT_MEMORY)
      if(i < MAX_SPHERES) {
         hit = SphereIntersectionTest(constSpheres[i], r, &h);
      } else {
         hit = SphereIntersectionTest(spheres[i], r, &h);
      }
#else
      hit = SphereIntersectionTest(spheres[i], r, &h);
#endif
      if(hit) {
         if (spheres[i].draw && h.depth < sh->depth) {
            *sh = h;
         }
      }
   }
}

#ifdef USE_SPHERE_TEST
__device__ data::color TraceLights(data::Ray &primary, data::Hit &hit, data::Sphere *spheres, int numSpheres, data::Pointlight *lights, int numLights, int *t) {
#else
__device__ data::color TraceLights(data::Ray &primary, data::Hit &hit, data::Sphere *spheres, int numSpheres, data::Pointlight *lights, int numLights) {
#endif
   data::color col(0,0,0);
   for(int i = 0; i < numLights; i++) {
      data::Hit sh;
      data::Ray r;
      r.start = hit.position;
      r.dir = (lights[i].position - hit.position).normalize();
      float distance = (lights[i].position - hit.position).distance();

#ifdef USE_BVH
 #ifdef USE_SPHERE_TEST
      cuda::CastRayBVH(r, spheres, &sh, t);
 #else
      cuda::CastRayBVH(r, spheres, &sh);
 #endif
#else
 #ifdef USE_SPHERE_TEST
      cuda::CastRayLinear(r, spheres, numSpheres, &sh, t);
 #else
      cuda::CastRayLinear(r, spheres, numSpheres, &sh);
 #endif
#endif

      if (sh.depth > distance) {
         col += lights[i].pColor * r.dir.dot(hit.normal);
      }
      //if(r.dir.dot(hit.normal) < 0) {
      //   col += lights[i].pColor * -r.dir.dot(hit.normal);
      //}
   }
   return col;
}


// This tests multiple spheres
#ifdef USE_SPHERE_TEST
__device__ data::color CastRay(data::Ray &r, data::Sphere *spheres, int numSpheres, data::Pointlight *lights, int numLights, int *t) {
#else
__device__ data::color CastRay(data::Ray &r, data::Sphere *spheres, int numSpheres, data::Pointlight *lights, int numLights) {
#endif
   data::Hit h;
   float prevDepth = h.depth;
#ifdef USE_SPHERE_TEST
   *t = 0;
#endif

#ifdef USE_BVH
 #ifdef USE_SPHERE_TEST
   cuda::CastRayBVH(r, spheres, &h, t);
 #else
   cuda::CastRayBVH(r, spheres, &h);
 #endif
#else
 #ifdef USE_SPHERE_TEST
   cuda::CastRayLinear(r, spheres, numSpheres, &h, t);
 #else
   cuda::CastRayLinear(r, spheres, numSpheres, &h);
 #endif
#endif
   if(h.depth < 1.0f) {
      return data::color(1.0f, 0, 0);
   } else if(h.depth < prevDepth) {
      // Create a ray for each light and test to see if it hits      
#ifdef USE_SPHERE_TEST
      return TraceLights(r, h, spheres, numSpheres, lights, numLights,t);
#else
      return TraceLights(r, h, spheres, numSpheres, lights, numLights);
#endif     
   } else {
      return data::color(0.0f, 0.0f, 0.0f);
   }
}



#ifdef USE_SPHERE_TEST
__global__ void RayTrace(data::Sphere *spheres, const int numSpheres,
                         data::Pointlight *lights, const int numLights, 
                         int *tests, uchar4 *pos, data::Camera c) {
#else
__global__ void RayTrace(data::Sphere *spheres, const int numSpheres,
                         data::Pointlight *lights, const int numLights, 
                         uchar4 *pos, data::Camera c) {
#endif
   int row = blockIdx.y*blockDim.y + threadIdx.y;
   int col = blockIdx.x*blockDim.x + threadIdx.x; // thread id (index to vectors a,b,c)

#ifdef ASSUME_ALL_CONST
   spheres = constSpheres;
#endif

   if(col < WIDTH && row < HEIGHT) {
      data::color clr;
      data::Ray r;

      r.start = c.position;
      float yaw = (float)(col - WIDTH/2)/(float)WIDTH * VIEW_ANGLE + (3.14159f * c.mYaw /180.f);
      float pitch = (float)(row - HEIGHT/2)/(float)HEIGHT * VIEW_ANGLE + (3.14159f * c.mPitch /180.f);
      data::vec3 dirmod(cos(yaw), sin(pitch), sin(yaw));
      r.dir = (data::vec3(0, 0, 0) + dirmod).normalize();


#ifdef USE_SPHERE_TEST
      clr = CastRay(r, spheres, numSpheres, lights, numLights, &tests[row*WIDTH + col]);
#else
      clr = CastRay(r, spheres, numSpheres, lights, numLights);
#endif

      // Each thread writes one pixel location in the texture (textel)
      pos[row*WIDTH + col].w = 0;
      pos[row*WIDTH + col].x = clr[0]*255;
      pos[row*WIDTH + col].y = clr[1]*255;
      pos[row*WIDTH + col].z = clr[2]*255;
   }
}

}

float rf() {
   return (float)rand()/(float)RAND_MAX;
}

void randomLights(data::Pointlight *l, int numLights) {
   for (int i = 0; i < numLights; i++) {
      float latitude = (rf() - 0.5f) * 3.14159f;
      float longitude = (rf() * 2.f * 3.14159f); 
      l[i].position = data::vec3(longitude, latitude, longitude);
      l[i].pColor = data::color(rf() + .3f,rf() + .3f,rf() + .3f);
      l[i].period = (2.f*rf() + 1)/90.f;
   }
}


data::Pointlight *cLights;
data::Sphere *cSpheres;
#ifdef USE_SPHERE_TEST
int *cTests;
#endif

const int numLights = 2;
data::Pointlight lights[numLights];
int numSpheres;

data::Camera cam;

#include <fstream>
#include <vector>
std::vector<data::vec3> randomPoints() {
   // 100 random spheres
   std::vector<data::vec3> points;
   for(int i = 0; i < 100; i++)
      points.push_back(data::vec3(rf(), rf(), rf())*10);
   return points;
}

std::vector<data::vec3> bunnyPoints() {
   // Read the points into an array
   int i = 0;
   char c;
   data::vec3 v;
   std::vector<data::vec3> points;
   std::ifstream iss("bunny.obj"); 
   iss >> c;
   while (c == 'v') {
      iss >> v.x >> v.y >> v.z;
      v *= 40;
      v.y *= -1;
      v += data::vec3(3, 0, 3);
      if (!(++i%3))
         points.push_back(v);
      iss >> c;
   }
   return points;
}

std::vector<data::vec3> linePoints() {
   std::vector<data::vec3> points;
   for(int i = 1; i < 100; i++)
      points.push_back(data::vec3(10,0,i*10));
   return points;
}

void randomSpheresBVH(data::Sphere **s, int *numSpheres) {
#ifdef USE_BUNNY
   std::vector<data::vec3> points = bunnyPoints();
#elif defined(USE_LINE_POINTS)
   std::vector<data::vec3> points = linePoints();
#else
   std::vector<data::vec3> points = randomPoints();
#endif

   // Sort the points according to their spatial location
   KDSort(&points[0], points.size());

   // Convert points into BVH spheres
   BVH<data::vec3, float>::Sphere sp;
   sp.radius = RADIUS;
   std::vector<BVH<data::vec3, float>::Sphere> spheres;
   for(std::vector<data::vec3>::iterator it = points.begin(); it != points.end(); it++) {
      sp.position = *it;
      spheres.push_back(sp);
   }
 
   // Create BVH
   BVH<data::vec3, float> bvh(&spheres[0], spheres.size());

   // Convert BVH spheres into renderable spheres
   *numSpheres = bvh.treeSize;
   *s = new data::Sphere[*numSpheres];
   for (int i = 0; i < *numSpheres; i++) {
      (*s)[i].center = bvh.mNodes[i].sphere.position;
      (*s)[i].radius = bvh.mNodes[i].sphere.radius;
      (*s)[i].draw = bvh.mNodes[i].isLeaf;
   }

}

void randomSpheresLinear(data::Sphere **s, int *numSpheres) {
#ifdef USE_BUNNY
   std::vector<data::vec3> points = bunnyPoints();
#elif defined(USE_LINE_POINTS)
   std::vector<data::vec3> points = linePoints();
#else
   std::vector<data::vec3> points = randomPoints();
#endif
   *numSpheres = points.size();
   *s = new data::Sphere[*numSpheres];
   for (int i = 0; i < *numSpheres; i++) {
      (*s)[i].center = points[i];
      (*s)[i].radius = RADIUS;
      (*s)[i].draw = true;
   }
}

extern void moveCamera(int direction) {
   cam.move(direction);
}

extern void printCamera() {
   cam.print();
}

extern void rotateCamera(int x, int y) {
   cam.rotate(x, y);
}

extern void init_kernel() {
#ifdef USE_SPHERE_TEST
   // Used to check the number of sphere tests we do
   HANDLE_ERROR(hipMalloc((void **) &cTests, HEIGHT*WIDTH*sizeof(int)));
#endif

   randomLights(&lights[0], numLights);
   HANDLE_ERROR(hipMalloc((void **) &cLights, numLights*sizeof(data::Pointlight)));
   HANDLE_ERROR(hipMemcpy(cLights, lights, numLights*sizeof(data::Pointlight), hipMemcpyHostToDevice));

   data::Sphere *spheres;
#ifdef USE_BVH
   randomSpheresBVH(&spheres, &numSpheres);
#else
   randomSpheresLinear(&spheres, &numSpheres);
#endif
   HANDLE_ERROR(hipMalloc((void **) &cSpheres, numSpheres*sizeof(data::Sphere)));
   HANDLE_ERROR(hipMemcpy(cSpheres, spheres, numSpheres*sizeof(data::Sphere), hipMemcpyHostToDevice));
#ifdef USE_CONSTANT_MEMORY
   HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda::constSpheres), spheres, min(numSpheres,MAX_SPHERES)*sizeof(data::Sphere)));
#endif
}


extern void destroy_kernel() {
   HANDLE_ERROR(hipFree(cLights));
   HANDLE_ERROR(hipFree(cSpheres));
#ifdef USE_SPHERE_TEST
   HANDLE_ERROR(hipFree(cTests));
#endif
}

int frame = 0;

// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel(uchar4* pos) {
   hipEvent_t start, stop;

   dim3 dimGrid(WIDTH/TILE_WIDTH + 1, HEIGHT/TILE_WIDTH + 1, 1);
   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

   frame++;
   data::Pointlight plights[numLights];
   for(int i = 0; i < numLights; i++) {
      plights[i].position.x = 30*sin(lights[i].position.x + frame * lights[i].period);
      plights[i].position.y = 30*sin(lights[i].position.y + frame * lights[i].period);
      plights[i].position.z = 30*cos(lights[i].position.z + frame * lights[i].period);
      plights[i].pColor = lights[i].pColor;
   }
   HANDLE_ERROR(hipMemcpy(cLights, plights, numLights*sizeof(data::Pointlight), hipMemcpyHostToDevice));

   HANDLE_ERROR(hipEventCreate(&start));
   HANDLE_ERROR(hipEventCreate(&stop));
   HANDLE_ERROR(hipEventRecord(start, 0));

#ifdef USE_SPHERE_TEST
   cuda::RayTrace<<<dimGrid, dimBlock>>>(cSpheres, numSpheres, cLights, numLights, cTests, pos, cam);
#else
   cuda::RayTrace<<<dimGrid, dimBlock>>>(cSpheres, numSpheres, cLights, numLights, pos, cam);
#endif

   HANDLE_ERROR(hipEventRecord(stop, 0));
   HANDLE_ERROR(hipEventSynchronize(stop));
   float elapsedTime;
   HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
   HANDLE_ERROR(hipEventDestroy(start));
   HANDLE_ERROR(hipEventDestroy(stop));

#ifdef USE_SPHERE_TEST
   static int tests[WIDTH * HEIGHT];
   HANDLE_ERROR(hipMemcpy(tests, cTests, HEIGHT*WIDTH*sizeof(int), hipMemcpyDeviceToHost));
   double numTests = 0;
   for(int i = 0; i < HEIGHT*WIDTH; i++) {
      numTests = numTests + tests[i];
   }
   numTests /= HEIGHT*WIDTH;
   printf("Time to generate: %.1f ms --- Average sphere intersection tests per ray: %.1f\n", elapsedTime, numTests);
#else
   printf("Time to generate: %.1f ms\n", elapsedTime);
#endif
   hipDeviceSynchronize();
}
